// I learn a lot from https://github.com/galeselee/taichi_benchmark/blob/prefix_sum/scan/src/cuda, which was written from me two years ago.
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>


#define TPB1D 1024

inline void cuAssert(hipError_t status, const char *file, int line) {
    if (status != hipSuccess)
        std::cerr<<"cuda assert: "<<hipGetErrorString(status)<<", file: "<<file<<", line: "<<line<<std::endl;
}
#define cuErrCheck(res)                                 \
    {                                                   \
        cuAssert((res), __FILE__, __LINE__);            \
    }

void Initialize(float *h_in, int num_items) {
    for (int ii = 0; ii < num_items; ii++) h_in[ii] = float(ii)/1000.0f;
}

void Solve(float *h_in, float *h_reference, int num_items) {
    float inclusive = 0;
    for (int ii = 0; ii < num_items; ii++) {
        inclusive += h_in[ii];
        h_reference[ii] = inclusive;
    }
    return ;
}

int assnear(float a, float b, float abs_err = 1e-1, float rel_err = 1e-3) {
    if(abs(a-b) > abs_err && abs(a-b)/a > rel_err) return 0;
    return 1;
}

void TestResult(float *h_out, float *h_reference, int nums) {
    for(int ii = 0; ii < nums; ii++) {
        if(!assnear(h_reference[ii], h_out[ii])) {
            // printf("FATAL : Error at %d : reference = %f, out = %f\n", ii, h_reference[ii], h_out[ii]);
            return;
        }
    }
}


__device__ float WarpScan(float val) {
    int lane = threadIdx.x & 31;
    float tmp = __shfl_up_sync(0xffffffff, val, 1);
    if (lane >= 1) val += tmp;
    tmp = __shfl_up_sync(0xffffffff, val, 2);
    if (lane >= 2) val += tmp;
    tmp = __shfl_up_sync(0xffffffff, val, 4);
    if (lane >= 4) val += tmp;
    tmp = __shfl_up_sync(0xffffffff, val, 8);
    if (lane >= 8) val += tmp;
    tmp = __shfl_up_sync(0xffffffff, val, 16);
    if (lane >= 16) val += tmp;
    __syncthreads();
    return val;
}

__device__ float BlockScan(float val) {
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;
    __shared__ float warp_sum[32];

    val = WarpScan(val);
    __syncthreads();
    if(lane == 31) warp_sum[warp_id] = val;
    __syncthreads();
    if(warp_id == 0) {
        if (lane >= 1) warp_sum[lane] += warp_sum[lane-1];
        __syncwarp();
        if (lane >= 2) warp_sum[lane] += warp_sum[lane-2];
        __syncwarp();
        if (lane >= 4) warp_sum[lane] += warp_sum[lane-4];
        __syncwarp();
        if (lane >= 8) warp_sum[lane] += warp_sum[lane-8];
        __syncwarp();
        if (lane >= 16) warp_sum[lane] += warp_sum[lane-16];
    }
    __syncthreads();
    if(warp_id > 0) val += warp_sum[warp_id-1];
    __syncthreads();
    return val;
}

__global__ 
void ScanKernel(float *in, float *out,
        float *buffer, int num_items, int num_part) {
    for(int ii = blockIdx.x; ii < num_part; ii += gridDim.x) {
        int idx = blockDim.x * ii + threadIdx.x;
        float val = idx < num_items ? in[idx] : 0;
        val = BlockScan(val);
        if(idx < num_items) out[idx] = val;
        if(threadIdx.x == blockDim.x - 1 && idx < num_items) {
            buffer[ii] = val;
        }
    }
}

__global__ void AddBaseKernel(float *buffer, float *out,
    int num_items, int num_part) {
    for(int ii = blockIdx.x; ii < num_part; ii += gridDim.x) {
        if(ii == 0) continue;
        int idx = ii * blockDim.x + threadIdx.x;
        if(idx < num_items) out[idx] += buffer[ii - 1];
    }
}

void Scan(float *d_in, float *d_out, float *buffer, int num_items) {
    int TPB = TPB1D;
    int num_part = (num_items + TPB - 1) / TPB;
    int BPG = std::min<int>(num_part, 256);
    ScanKernel<<<BPG, TPB>>> (
        d_in, d_out, buffer, num_items, num_part);
    if(num_part >= 2) {
        Scan(buffer, buffer + num_part, buffer, num_part);
        AddBaseKernel<<<BPG, TPB>>>(buffer+num_part, d_out, num_items, num_part);
    }
}

int main(int argc, char **argv) {
    int num_items = 4096;
    if(argc > 1) num_items = std::atoi(argv[1]);
    float *d_in = nullptr;
    float *d_out = nullptr;
    float *buffer = nullptr;
    float *h_in = new float [num_items];
    float *h_out = new float [num_items];
    float *h_reference = new float [num_items];

    Initialize(h_in, num_items);
    Solve(h_in, h_reference, num_items);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_in, num_items * sizeof(float));
    hipMalloc(&d_out, num_items * sizeof(float));
    // Loose array
    hipMalloc(&buffer, (num_items + TPB1D - 1) / TPB1D * 4 * sizeof(float));
    hipMemset((void *)buffer, 0, (num_items + TPB1D - 1) / TPB1D * 4 * sizeof(float));
    
    cuErrCheck(hipMemcpy(d_in, h_in, sizeof(float) * num_items, hipMemcpyHostToDevice));
    
    hipEventRecord(start);

    Scan(d_in, d_out, buffer, num_items);
    hipDeviceSynchronize();
    cuErrCheck(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f", milliseconds);

    cuErrCheck(hipMemcpy(h_out, d_out, sizeof(float) * (num_items), hipMemcpyDeviceToHost));
    TestResult(h_out, h_reference, num_items);

    hipFree(d_in);
    hipFree(buffer);
    hipFree(d_out);
    delete[] h_in;
    delete[] h_reference;
    delete[] h_out;
    return 0;
}